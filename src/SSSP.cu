#include <cstdint>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <vector>

#include "Algorithms/SSSP.cuh"

int main(int argc, char *argv[]) {
  using AllocPolicy = FullAllocatorPolicy<9, 48, 1>;
  using Allocator = typename AllocPolicy::DynamicAllocatorT;

  std::cout << argv[1];
  std::flush(std::cout);

  std::string Filename = std::string(argv[1]);
  std::string Op = std::string(argv[2]);
  std::string UpdateFile = std::string(argv[3]);
  std::string NumBatches = std::string(argv[4]);
  uint32_t Seed = std::atoi(argv[5]);

  std::srand(Seed);

  bool IsWikiTalk = (std::strstr(Filename.c_str(), "wiki.txt") != NULL);
  bool IsOrkut = (std::strstr(Filename.c_str(), "orkut.txt") != NULL);
  uint32_t TheSrc = IsWikiTalk ? 2 : 0;

  std::FILE *GraphFile = std::fopen(Filename.c_str(), "r");
  uint32_t VertexN, EdgeN;
  float ElapsedTime = 0.0f;

  std::fscanf(GraphFile, "%d%d", &VertexN, &EdgeN);

  // initialising degree hints with 1
  std::vector<uint32_t> VertexDegreeHints{
      std::move(std::vector<uint32_t>(VertexN, 1))};
  std::unique_ptr<uint32_t[]> Src{new uint32_t[EdgeN]};
  std::unique_ptr<uint32_t[]> Dst{new uint32_t[EdgeN]};
  std::unique_ptr<uint32_t[]> Wgt{new uint32_t[EdgeN]};

  if (IsOrkut) {
    for (int i = 0; i < EdgeN; ++i) {
      (void)fscanf(GraphFile, "%d%d", &Src[i], &Dst[i]);
      Wgt[i] = 1 + (rand() % 100);
      ++VertexDegreeHints[Src[i]];
    }
  } else {
    for (int i = 0; i < EdgeN; ++i) {
      (void)fscanf(GraphFile, "%d%d", &Src[i], &Dst[i]);
      Wgt[i] = 1 + (rand() % 100);
    }
  }

  double LoadFactor = 0.7;

  uint32_t *SrcDevPtr;
  uint32_t *DstDevPtr;
  uint32_t *WgtDevPtr;

  // initialise GPU variables and memory
  hipSetDevice(0);
  CHECK_ERROR(hipMalloc(&SrcDevPtr, sizeof(uint32_t) * EdgeN));
  CHECK_ERROR(hipMalloc(&DstDevPtr, sizeof(uint32_t) * EdgeN));
  CHECK_ERROR(hipMalloc(&WgtDevPtr, sizeof(uint32_t) * EdgeN));

  CHECK_ERROR(hipMemcpy(SrcDevPtr, Src.get(), sizeof(uint32_t) * EdgeN,
                         hipMemcpyHostToDevice));
  CHECK_ERROR(hipMemcpy(DstDevPtr, Dst.get(), sizeof(uint32_t) * EdgeN,
                         hipMemcpyHostToDevice));
  CHECK_ERROR(hipMemcpy(WgtDevPtr, Wgt.get(), sizeof(uint32_t) * EdgeN,
                         hipMemcpyHostToDevice));

  using AllocPolicy = FullAllocatorPolicy<9, 48, 1>;
  using Allocator = typename AllocPolicy::DynamicAllocatorT;
  std::unique_ptr<Allocator> Alloc{new Allocator};

  using DynGraph = typename SSSP<AllocPolicy>::GraphT;
  using DynGraphContext = typename DynGraph::GraphContextT;
  DynGraph Graph{VertexN, *Alloc, LoadFactor, VertexDegreeHints.data(), 0};

  Graph.InsertEdges(SrcDevPtr, DstDevPtr, EdgeN, WgtDevPtr);
  hipDeviceSynchronize();

  CHECK_ERROR(hipFree(SrcDevPtr));
  CHECK_ERROR(hipFree(DstDevPtr));
  CHECK_ERROR(hipFree(WgtDevPtr));

  SSSP<AllocPolicy> S(Graph, TheSrc, VertexN);
  ElapsedTime = S.Static(EdgeN);

  std::cout << "," << ElapsedTime;
  std::flush(std::cout);

  bool IsInsertion = (Op == "0");
  uint32_t BatchesN = std::stoi(NumBatches);

  for (int File = 0; File < BatchesN; ++File) {
    std::FILE *BatchFile = std::fopen(
        std::string(UpdateFile + ".batch." + std::to_string(File)).c_str(),
        "r");
    uint32_t BatchEdgesN;
    std::fscanf(BatchFile, "%d", &BatchEdgesN);

    uint32_t *BatchEdgesSrc = new uint32_t[BatchEdgesN];
    uint32_t *BatchEdgesDst = new uint32_t[BatchEdgesN];
    uint32_t *BatchEdgesWgt = new uint32_t[BatchEdgesN];
    uint32_t Src, Dst;

    for (uint32_t I = 0; I < BatchEdgesN; ++I) {
      std::fscanf(BatchFile, "%d%d", &Src, &Dst);
      BatchEdgesSrc[I] = Src;
      BatchEdgesDst[I] = Dst;
      BatchEdgesWgt[I] = 1 + (rand() % 100);
    }

    uint32_t *BatchEdgesSrcDev;
    uint32_t *BatchEdgesDstDev;
    uint32_t *BatchEdgesWgtDev;

    CHECK_ERROR(hipMalloc(&BatchEdgesSrcDev, sizeof(uint32_t) * BatchEdgesN));
    CHECK_ERROR(hipMemcpy(BatchEdgesSrcDev, BatchEdgesSrc,
                           sizeof(uint32_t) * BatchEdgesN,
                           hipMemcpyHostToDevice));

    CHECK_ERROR(hipMalloc(&BatchEdgesDstDev, sizeof(uint32_t) * BatchEdgesN));
    CHECK_ERROR(hipMemcpy(BatchEdgesDstDev, BatchEdgesDst,
                           sizeof(uint32_t) * BatchEdgesN,
                           hipMemcpyHostToDevice));

    CHECK_ERROR(hipMalloc(&BatchEdgesWgtDev, sizeof(uint32_t) * BatchEdgesN));
    CHECK_ERROR(hipMemcpy(BatchEdgesWgtDev, BatchEdgesWgt,
                           sizeof(uint32_t) * BatchEdgesN,
                           hipMemcpyHostToDevice));

    delete[] BatchEdgesSrc;
    delete[] BatchEdgesDst;
    delete[] BatchEdgesWgt;

    bool IsInsertion = (Op == "0");

    if (IsInsertion)
      Graph.InsertEdges(BatchEdgesSrcDev, BatchEdgesDstDev, BatchEdgesN,
                        BatchEdgesWgtDev);
    else
      Graph.DeleteEdges(BatchEdgesSrcDev, BatchEdgesDstDev, BatchEdgesN);
    hipDeviceSynchronize();

    float ElapsedTime;
    ElapsedTime = IsInsertion
                      ? S.Incremental(BatchEdgesSrcDev, BatchEdgesDstDev,
                                      BatchEdgesWgtDev, BatchEdgesN)
                      : S.Decremental(BatchEdgesSrcDev, BatchEdgesDstDev,
                                      BatchEdgesWgtDev, BatchEdgesN);

    CHECK_ERROR(hipFree(BatchEdgesSrcDev));
    CHECK_ERROR(hipFree(BatchEdgesDstDev));
    CHECK_ERROR(hipFree(BatchEdgesWgtDev));

    std::cout << "," << ElapsedTime;
    std::flush(std::cout);
  }

  std::cout << std::endl;
  return 0;
}